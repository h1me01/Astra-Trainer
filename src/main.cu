#include "hip/hip_runtime.h"
#include "nn/network.h"

using namespace std;

int main() {
    const string root_path = "D:/Astra-Data";

    cout << "================================= Training Data ================================\n\n";

    // get training data
    vector<string> files = fetchFilesFromPath(root_path + "/training_data");

    // init network
    Network network( //
        800,         // epochs
        16384,       // batch size
        6104,        // batches per epoch
        100,         // save rate
        1,           // thread count for dataloader
        400,         // output scalar
        1.0,         // start lambda
        1.0          // end lambda
    );

    // init loss
    MPELoss<Sigmoid> loss(2.5); // 2.5 = power
    network.setLoss(&loss);

    // init optim
    Adam optim( //
        0.001,  // lr
        0.9,    // beta1
        0.999,  // beta2
        1e-8    // epsilon
    );

    // init lr scheduler
    StepDecay lr_sched( //
        160,            // step size
        0.1             // gamma
    );

    // GradualDecay lr_sched(0.99); // 0.99 = gamma

    // CosineAnnealing lr_sched(   //
    //     network.getBatchSize(), // max epochs
    //     0.001,                  // lr
    //     0.001f * powf(0.3f, 5)  // min lr
    //);

    optim.setDecay(0.01);
    optim.setLRScheduler(&lr_sched);
    optim.clamp(-1.99, 1.99); // all weights & biases range [-1.99, 1.99]

    network.setOptimizer(&optim);

    // init king bucket (if needed)
    array<int, 64> king_bucket = {
        0,  1,  2,  3,  3,  2,  1,  0,  //
        4,  5,  6,  7,  7,  6,  5,  4,  //
        8,  8,  9,  9,  9,  9,  8,  8,  //
        10, 10, 10, 10, 10, 10, 10, 10, //
        10, 10, 10, 10, 10, 10, 10, 10, //
        11, 11, 11, 11, 11, 11, 11, 11, //
        11, 11, 11, 11, 11, 11, 11, 11, //
        11, 11, 11, 11, 11, 11, 11, 11, //
    };

    network.setKingBucket(king_bucket);

    // init hidden layers
    auto ft = FeatureTransformer<256, SCReLU>(getBucketSize(king_bucket) * 768);
    auto fc = FullyConnected<1, Linear>(&ft);

    network.setHiddenLayers({&ft, &fc});

    // setup quantization scheme
    network.setQuantizationScheme([&](FILE *f) {
        const int q1 = 255;
        const int q2 = 64;

        ft.getParams()[0]->quantize<int16_t>(f, q1, true); // weights
        ft.getParams()[1]->quantize<int16_t>(f, q1);       // biases
        fc.getParams()[0]->quantize<int16_t>(f, q2);       // weights
        fc.getParams()[1]->quantize<int16_t>(f, q1 * q2);  // biases
    });

    const string output_path = root_path + "/nn_output";

    // load weights only (if needed)
    // network.loadWeights(output_path + "/training_3/checkpoint-final/weights.bin");
    network.train(  //
        files,      //
        output_path //
                    // "training_4/checkpoint-100" // load checkpoint (if needed)
    );

    cout << "\n================================ Testing Network ===============================\n\n";

    vector<string> test_fens = {
        "rnbqkbnr/pppppppp/8/8/8/8/PPPPPPPP/RNBQKBNR w KQkq - 0 1",
        "rn1qk2r/ppp1bppp/5n2/3p1bB1/3P4/2N1P3/PP3PPP/R2QKBNR w KQkq - 1 7",
    };

    for(auto fen : test_fens) {
        cout << "FEN: " << fen << endl;
        cout << "Eval: " << network.predict(fen) << endl;
    }

    cout << "\n=================================== Finished ===================================\n";
}
