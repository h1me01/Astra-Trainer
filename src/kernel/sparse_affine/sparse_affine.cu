#include "hip/hip_runtime.h"
#include "sparse_affine.h"

__global__ void sparse_affine_kernel( //
    const float *weights_v,
    const float *biases_v,
    float *activated_v,
    float *prev_activated,
    const int *features,
    const int *feature_sizes,
    const int w_r,      // weight rows
    const int a_r,      // activated rows
    const int a_offset, // activated offset
    const int batch_size,
    const int max_entries,
    ActivationType act_type) //
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= w_r * batch_size)
        return;

    const int batch_idx = idx / w_r;
    const int neuron_idx = idx % w_r;

    const int offset = batch_idx * max_entries;
    const int feature_size = feature_sizes[batch_idx];

    float sum = biases_v[neuron_idx];
    for(int i = 0; i < feature_size; i++) {
        int sparse_idx = features[i + offset];
        sum += weights_v[w_r * sparse_idx + neuron_idx];
    }

    int output_idx = a_r * batch_idx + neuron_idx + a_offset;

    prev_activated[output_idx] = sum;
    activated_v[output_idx] = activate(sum, act_type);
}

__global__ void sparse_affine_bp_kernel( //
    const float *activated_g,
    const float *prev_activated,
    float *weights_g,
    float *biases_g,
    const int *features,
    const int *feature_sizes,
    const int w_r,      // weight rows
    const int a_r,      // activated rows
    const int a_offset, // activated offset
    const int batch_size,
    const int max_entries,
    ActivationType act_type) //
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= w_r * batch_size)
        return;

    const int batch_idx = idx / w_r;
    const int neuron_idx = idx % w_r;

    const int output_idx = a_r * batch_idx + neuron_idx + a_offset;

    float grad = activated_g[output_idx];
    if(grad == 0)
        return;
    grad *= activationDer(prev_activated[output_idx], act_type);

    // no need to compute gradients for previous layer since previous are inputs

    const int offset = batch_idx * max_entries;
    const int feature_size = feature_sizes[batch_idx];

    atomicAdd(&biases_g[neuron_idx], grad);
    for(int i = 0; i < feature_size; i++) {
        int sparse_idx = features[i + offset];
        atomicAdd(&weights_g[w_r * sparse_idx + neuron_idx], grad);
    }
}
