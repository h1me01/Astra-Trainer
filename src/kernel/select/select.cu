#include "hip/hip_runtime.h"
#include "select.h"

const int block_size = 128;

// FORWARD

__global__ void select_fwd_kernel( //
    const float *input_v,          //
    float *output_v,               //
    const int *bucket_indices,     //
    const int batch_size,          //
    const int input_size,          //
    const int output_size          //
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= batch_size * output_size)
        return;

    const int batch_idx = idx / output_size;
    const int output_idx = idx % output_size;

    const int bucket = bucket_indices[batch_idx];
    const int input_offset = input_size * batch_idx + output_size * bucket + output_idx;

    output_v[output_size * batch_idx + output_idx] = input_v[input_offset];
}

void select_fwd(                       //
    const DenseMatrix<float> &input_v, //
    DenseMatrix<float> &output_v,      //
    const Array<int> &bucket_indices,  //
    const int batch_size,              //
    const int input_size,              //
    const int output_size              //
) {
    const int grid_size = std::ceil(batch_size * output_size / block_size);

    select_fwd_kernel<<<grid_size, block_size>>>( //
        input_v.dev_address(),
        output_v.dev_address(),
        bucket_indices.dev_address(),
        batch_size,
        input_size,
        output_size);
}

// BACKWARD

__global__ void select_bwd_kernel( //
    float *input_g,                //
    const float *output_g,         //
    const int *indices,            //
    const int batch_size,          //
    const int input_size,          //
    const int output_size          //
) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= batch_size * output_size)
        return;

    const int batch_idx = idx / output_size;
    const int output_idx = idx % output_size;

    const int bucket = indices[batch_idx];
    const int input_offset = input_size * batch_idx + output_size * bucket + output_idx;

    input_g[input_offset] = output_g[output_size * batch_idx + output_idx];
}

void select_bwd(                        //
    DenseMatrix<float> &input_g,        //
    const DenseMatrix<float> &output_g, //
    const Array<int> &indices,          //
    const int batch_size,               //
    const int input_size,               //
    const int output_size               //
) {
    ASSERT(batch_size == indices.size());

    const int grid_size = std::ceil(batch_size * output_size / block_size);

    // clear input gradient
    input_g.clear_dev();

    select_bwd_kernel<<<grid_size, block_size>>>( //
        input_g.dev_address(),
        output_g.dev_address(),
        indices.dev_address(),
        batch_size,
        input_size,
        output_size);
}