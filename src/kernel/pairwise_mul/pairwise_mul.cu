#include "hip/hip_runtime.h"
#include "pairwise_mul.h"

constexpr int block_size = 1024;

// FORWARD

__global__ void pairwise_mul_fwd_kernel( //
    const float *inputs_v,               //
    float *output_v,                     //
    const int output_size,               //
    const int batch_size                 //
) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= output_size * batch_size)
        return;

    const int batch_idx = idx / output_size;
    const int output_idx = idx % output_size;

    const int input_offset = 2 * output_size * batch_idx + output_idx;
    const int output_offset = output_size * batch_idx + output_idx;

    output_v[output_offset] = inputs_v[input_offset] * inputs_v[input_offset + output_size];
}

void pairwise_mul_fwd(                  //
    const DenseMatrix<float> &inputs_v, //
    DenseMatrix<float> &output_v        //
) {
    ASSERT(inputs_v.is_dev_allocated() //
           && output_v.is_dev_allocated());

    const int batch_size = output_v.cols();
    const int output_size = output_v.rows();

    const int grid_size = std::ceil(float(output_size * batch_size) / block_size);

    pairwise_mul_fwd_kernel<<<grid_size, block_size>>>( //
        inputs_v.dev_address(),                         //
        output_v.dev_address(),                         //
        output_size,
        batch_size //
    );
}

// BACKWARD

__global__ void pairwise_mul_bwd_kernel( //
    const float *inputs_v,               //
    float *inputs_g,                     //
    const float *output_g,               //
    const int output_size,               //
    const int batch_size                 //
) {
    const int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx >= output_size * batch_size)
        return;

    const int batch_idx = idx / output_size;
    const int output_idx = idx % output_size;

    const int output_offset = output_size * batch_idx + output_idx;
    const int input_offset = 2 * output_size * batch_idx + output_idx;

    const float gradIn = output_g[output_offset];

    inputs_g[input_offset] += gradIn * inputs_v[input_offset + output_size];
    inputs_g[input_offset + output_size] += gradIn * inputs_v[input_offset];
}

void pairwise_mul_bwd(                //
    Tensor &inputs,                   //
    const DenseMatrix<float> output_g //
) {
    const DenseMatrix<float> &inputs_v = inputs.get_data();
    DenseMatrix<float> &inputs_g = inputs.get_grads();

    ASSERT(inputs_v.is_dev_allocated()    //
           && inputs_g.is_dev_allocated() //
           && output_g.is_dev_allocated());

    const int batch_size = output_g.cols();
    const int output_size = output_g.rows();

    const int grid_size = std::ceil(float(output_size * batch_size) / block_size);

    pairwise_mul_bwd_kernel<<<grid_size, block_size>>>( //
        inputs_v.dev_address(),                         //
        inputs_g.dev_address(),                         //
        output_g.dev_address(),                         //
        output_size,                                    //
        batch_size                                      //
    );
}
