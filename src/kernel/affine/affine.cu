#include "hip/hip_runtime.h"
#include "affine.h"

constexpr float alpha = 1;
constexpr float beta = 0;

// hipblasSgemm performs C = A * B * alpha + C * beta

hipblasHandle_t CUBLAS_HANDLE;

void createCublas() {
    hipblasCreate(&CUBLAS_HANDLE);
}

void destroyCublas() {
    hipblasDestroy(CUBLAS_HANDLE);
}

// AFFINE
__global__ void add_biases_kernel( //
    const float *biases_v,
    float *activated_v,
    float *pre_activated_v,
    const int r,
    const int c,
    const ActivationType act_type //
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= r * c)
        return;

    int neuron_idx = idx / c;

    float weighted_sum = pre_activated_v[idx] + biases_v[neuron_idx];

    pre_activated_v[idx] = weighted_sum;
    activated_v[idx] = activate(weighted_sum, act_type);
}

void affine( //
    DenseMatrix &weights_v,
    DenseMatrix &biases_v,
    DenseMatrix &inputs_v,
    DenseMatrix &activated_v,
    DenseMatrix &pre_activated,
    const ActivationType act_type //
) {
    ASSERT(activated_v.numRows() == biases_v.numRows() && biases_v.numCols() == 1);

    ASSERT(weights_v.numCols() == inputs_v.numRows() &&    //
           weights_v.numRows() == activated_v.numRows() && //
           inputs_v.numCols() == activated_v.numCols());

    ASSERT(weights_v.devAddress() &&   //
           biases_v.devAddress() &&    //
           inputs_v.devAddress() &&    //
           activated_v.devAddress() && //
           pre_activated.devAddress());

    // compute dot product
    hipblasSgemm(                    //
        CUBLAS_HANDLE,              // handle
        HIPBLAS_OP_N,                // transa
        HIPBLAS_OP_N,                // transb
        pre_activated.numRows(),    // m
        pre_activated.numCols(),    // n
        weights_v.numCols(),        // k
        &alpha,                     // alpha
        weights_v.devAddress(),     // A
        weights_v.numRows(),        // lda
        inputs_v.devAddress(),      // B
        inputs_v.numRows(),         // ldb
        &beta,                      // beta
        pre_activated.devAddress(), // C
        pre_activated.numRows()     // ldc
    );

    // add biases to dot product
    const int block_size = 128;
    const int grid_size = std::ceil((float) activated_v.size() / block_size);

    add_biases_kernel<<<grid_size, block_size>>>( //
        biases_v.devAddress(),
        activated_v.devAddress(),
        pre_activated.devAddress(),
        activated_v.numRows(),
        activated_v.numCols(),
        act_type);
}

// AFFINE BP
__global__ void update_biases_grad_kernel( //
    const float *pre_activated_v,
    float *activated_g,
    float *biases_g,
    const int r,
    const int c,
    const ActivationType act_type //
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= r * c)
        return;

    float grad = activated_g[idx];
    if(grad == 0)
        return;

    grad *= activationDer(pre_activated_v[idx], act_type);
    activated_g[idx] = grad;

    int neuron_idx = idx / c;
    atomicAdd(&biases_g[neuron_idx], grad);
}

void affine_bp( //
    Tensor &weights,
    Tensor &biases,
    Tensor &inputs,
    Tensor &activated,
    DenseMatrix &pre_activated,
    const ActivationType act_type //
) {
    const DenseMatrix &weights_v = weights.getValues();
    DenseMatrix &weights_g = weights.getGradients();

    DenseMatrix &biases_g = biases.getGradients();

    const DenseMatrix &inputs_v = inputs.getValues();
    DenseMatrix &inputs_g = inputs.getGradients();

    const DenseMatrix &activated_v = activated.getValues();
    const DenseMatrix &activated_g = activated.getGradients();

    ASSERT(activated_g.numRows() == biases_g.numRows() && biases_g.numCols() == 1);

    ASSERT(weights_g.numCols() == inputs_g.numRows() &&    //
           weights_g.numRows() == activated_g.numRows() && //
           inputs_g.numCols() == activated_g.numCols());

    ASSERT(weights_v.devAddress() &&   //
           weights_g.devAddress() &&   //
           biases_g.devAddress() &&    //
           inputs_v.devAddress() &&    //
           inputs_g.devAddress() &&    //
           activated_v.devAddress() && //
           activated_g.devAddress() && //
           pre_activated.devAddress());

    // update biases gradient
    const int block_size = 128;
    const int grid_size = std::ceil((float) activated_g.size() / block_size);

    update_biases_grad_kernel<<<grid_size, block_size>>>( //
        pre_activated.devAddress(),
        activated_g.devAddress(),
        biases_g.devAddress(),
        activated_g.numRows(),
        activated_g.numCols(),
        act_type);

    // update weights gradient
    hipblasSgemm(                  //
        CUBLAS_HANDLE,            // handle
        HIPBLAS_OP_N,              // transa
        HIPBLAS_OP_T,              // transb
        weights_g.numRows(),      // m
        weights_g.numCols(),      // n
        activated_g.numCols(),    // k
        &alpha,                   // alpha
        activated_g.devAddress(), // A
        activated_g.numRows(),    // lda
        inputs_v.devAddress(),    // B
        inputs_v.numRows(),       // ldb
        &beta,                    // beta
        weights_g.devAddress(),   // C
        weights_g.numRows()       // ldc
    );

    // calculates delta for the layer before this one as well
    hipblasSgemm(                  //
        CUBLAS_HANDLE,            // handle
        HIPBLAS_OP_T,              // transa
        HIPBLAS_OP_N,              // transb
        inputs_g.numRows(),       // m
        inputs_g.numCols(),       // n
        weights_v.numRows(),      // k
        &alpha,                   // alpha
        weights_v.devAddress(),   // A
        weights_v.numRows(),      // lda
        activated_g.devAddress(), // B
        activated_g.numRows(),    // ldb
        &beta,                    // beta
        inputs_g.devAddress(),    // C
        inputs_g.numRows()        // ldc
    );
}
