#include "affine.h"

// hipblasSgemm performs C = A * B * alpha + C * beta

hipblasHandle_t CUBLAS_HANDLE;

void create_cublas() {
    hipblasCreate(&CUBLAS_HANDLE);
}

void destroy_cublas() {
    hipblasDestroy(CUBLAS_HANDLE);
}
