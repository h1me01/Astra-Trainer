#include "hip/hip_runtime.h"
#include "../misc.h"
#include "network.h"

int Network::index(PieceType pt, Color pc, Square psq, Square ksq, Color view) {
    int _psq = int(psq);
    int _ksq = int(ksq);
    int _pc = int(pc);
    int _pt = int(pt);
    int _view = int(view);

    const int ksIndex = king_bucket[(56 * _view) ^ _ksq];
    // relative square
    _psq = _psq ^ (56 * _view);
    // horizontal flip if king is on other half
    _psq = _psq ^ (7 * !!(_ksq & 0x4));

    return _psq + _pt * 64 + (_pc != _view) * 64 * 6 + ksIndex * 768;
}

void Network::fill(std::vector<DataEntry> &ds) {
    SparseBatch &sparse_inputs = layers[0]->getSparseBatch();

    const int max_entries = sparse_inputs.maxEntries();

    auto &stm_features = sparse_inputs.getFeatures()[0];
    auto &nstm_features = sparse_inputs.getFeatures()[1];
    auto &features_sizes = sparse_inputs.getFeatureSizes();

    for(size_t i = 0; i < ds.size(); i++) {
        const auto pos = ds[i].pos;

        auto ksq_w = pos.kingSquare(Color::White);
        auto ksq_b = pos.kingSquare(Color::Black);

        bool wtm = pos.sideToMove() == Color::White;
        auto pieces = pos.piecesBB();

        int offset = i * max_entries;
        int count = 0;
        for(auto sq : pieces) {
            auto p = pos.pieceAt(sq);
            auto w_idx = index(p.type(), p.color(), sq, ksq_w, Color::White);
            auto b_idx = index(p.type(), p.color(), sq, ksq_b, Color::Black);

            int idx = offset + count;
            stm_features(idx) = wtm ? w_idx : b_idx;
            nstm_features(idx) = wtm ? b_idx : w_idx;

            count++;
        }

        features_sizes(i) = count;

        float score_target = 1.0f / (1.0f + expf(-float(ds[i].score) / OutputScalar));
        float wdl_target = (ds[i].result + 1) / 2.0f;

        float actual_lambda = StartLambda + (EndLambda - StartLambda) * (epoch / float(Epochs));
        targets(i) = actual_lambda * score_target + (1.0f - actual_lambda) * wdl_target;
    }

    // upload to device
    targets.hostToDev();
    features_sizes.hostToDev();
    stm_features.hostToDev();
    nstm_features.hostToDev();
}

void Network::train(std::vector<std::string> &files, std::string output_path, std::string checkpoint_name) {
    init();
    printInfo();

    if(checkpoint_name.empty()) {
        std::cout << "No checkpoint path provided, training from scratch.\n";
    } else {
        std::cout << "Loading checkpoint from " << checkpoint_name << " ..." << std::endl;
        const std::string checkpoint_path = output_path + "/" + checkpoint_name;
        if(!std::filesystem::exists(checkpoint_path)) {
            std::cerr << "Checkpoint path does not exist: " << checkpoint_path << "\n";
            return;
        }

        loadWeights(checkpoint_path + "/weights.bin");
        optim->load(checkpoint_path);
    }

    // init dataloader
    FeaturedBatchStream dataloader(files, 4, BatchSize, false);

    std::cout << "\n=============================== Training Network ===============================\n\n";

    // add new folder to output path
    int next_training_index = getNextTrainingIndex(output_path);

    std::stringstream new_folder_path;
    new_folder_path << output_path << "/training_" << next_training_index;

    std::filesystem::create_directory(new_folder_path.str());
    std::cout << "Created folder: " << new_folder_path.str() << "\n\n";

    // save network info
    std::ofstream info_file(new_folder_path.str() + "/info.txt");
    if(info_file.is_open()) {
        info_file << info.str();
        info_file << dataloader.getInfo() << "\n";
        info_file.close();
    } else {
        std::cerr << "Failed to save info file!\n";
        return;
    }

    Logger log{new_folder_path.str() + "/loss.csv"};
    log.write({"epoch", "training_loss"});

    Timer timer;
    for(epoch = 1; epoch <= Epochs; epoch++) {
        timer.start();
        loss->reset();

        for(int batch = 1; batch <= BatchesPerEpoch; batch++) {
            auto ds = dataloader.next();
            fill(ds);

            timer.stop();
            auto elapsed = timer.getElapsedTime();

            if(batch == BatchesPerEpoch || timer.isTimeReached(1000)) {
                printf("\repoch/batch = %3d/%4d, ", epoch, batch);
                printf("pos/s = %7d, ", (int) round(1000.0f * BatchSize * batch / elapsed));
                printf("time = %3ds", (int) elapsed / 1000);
                std::cout << std::flush;
            }

            forward();
            loss->apply(targets, getOutput());
            backprop();
            optim->apply(ds.size());
        }

        float epoch_loss = loss->getLoss() / (BatchSize * BatchesPerEpoch);

        timer.stop();
        auto elapsed = timer.getElapsedTime();

        printf("\repoch/batch = %3d/%4d, ", epoch, BatchesPerEpoch);
        printf("loss = %1.8f, ", epoch_loss);
        printf("pos/s = %7d, ", (int) round(1000.0f * BatchSize * BatchesPerEpoch / elapsed));
        printf("time = %3ds", (int) elapsed / 1000);
        std::cout << std::endl;

        if(epoch % SaveRate == 0 || epoch == Epochs) {
            log.write({std::to_string(epoch), std::to_string(epoch_loss)});

            std::string suffix = epoch == Epochs ? "final" : std::to_string(epoch);
            saveCheckpoint(new_folder_path.str() + "/checkpoint-" + suffix);
        }

        optim->updateLR(epoch);
    }
}
